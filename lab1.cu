
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <iomanip>
#include <chrono>
#include <random>

void fill_random_array(double* arr, int n, double min_val, double max_val) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> distrib(min_val, max_val);

    for (int i = 0; i < n; ++i) {
        arr[i] = distrib(gen);
    }
}

__global__ void min_elements (double *arr_1, double *arr_2, double *res, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = gridDim.x * blockDim.x;

    while (idx < n) {
        if (arr_1[idx] < arr_2[idx]) {
            res[idx] = arr_1[idx];
        } else {
            res[idx] = arr_2[idx];
        }

        idx += offset;
    }
}

void min_elements_cpu (double *arr_1, double *arr_2, double *res, int n) {
    for (int idx = 0; idx < n; ++idx) {
        if (arr_1[idx] < arr_2[idx]) {
            res[idx] = arr_1[idx];
        } else {
            res[idx] = arr_2[idx];
        }
    }
}

int main () {
    std::ios::sync_with_stdio(false);
    std::cin.tie(nullptr);

    std::cout << std::fixed;
    std::cout << std::setprecision(10);

    int n;
//    n = 1000;

    std::cin >> n;

    if (n < 0) {
        return 0;
    }

    double *arr_1 = new double[n];
    double *arr_2 = new double[n];
    double *res = new double[n];

    for (int i = 0; i < n; ++i) {
        std::cin >> arr_1[i];
    }

    for (int i = 0; i < n; ++i) {
        std::cin >> arr_2[i];
    }

//    fill_random_array(arr_1, n, -1000, 1000);
//    fill_random_array(arr_2, n, -1000, 1000);

    double *dev_arr_1, *dev_arr_2, *dev_res;
    hipMalloc((void**)&dev_arr_1, sizeof(double) * n);
    hipMalloc((void**)&dev_arr_2, sizeof(double) * n);
    hipMalloc((void**)&dev_res, sizeof(double) * n);

    hipMemcpy(dev_arr_1, arr_1, sizeof(double) * n, hipMemcpyHostToDevice);
    hipMemcpy(dev_arr_2, arr_2, sizeof(double) * n, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    min_elements<<<1, 32>>>(dev_arr_1, dev_arr_2, dev_res, n);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

//    std::cout << "Time: " << elapsedTime << " ms" << std::endl;

    hipMemcpy(res, dev_res, sizeof(double) * n, hipMemcpyDeviceToHost);

    for (int i = 0; i < n; ++i) {
        std::cout << res[i] << " ";
    }

    delete [] arr_1;
    delete [] arr_2;
    delete [] res;
    hipFree(dev_arr_1);
    hipFree(dev_arr_2);
    hipFree(dev_res);

    return 0;
}